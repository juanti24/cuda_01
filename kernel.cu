
#include <hip/hip_runtime.h>
#include<iostream>

__global__
void addVecKernel(float* a, float* b, float* c, int size) {
    int index = threadIdx.x+blockDim.x*blockIdx.x;

    if(index<size){
        c[index]= a[index]+ b[index];
    }
}

extern "C"
void addVec(float* a, float* b, float* c, int size) {

    int thread_num = 1024;
    int num_bloques = std::ceil(size/thread_num);
    printf("num_bloques=%d, num_threads=%d\n",num_bloques,thread_num);
    addVecKernel<<<num_bloques,thread_num>>>(a,b,c,size);
}
